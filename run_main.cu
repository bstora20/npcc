#include "hip/hip_runtime.h"
__global__ static void run(struct Cell *pond, uintptr_t *buffer, int *in, uint64_t *prngState) 
{
    //const uintptr_t threadNo = (uintptr_t)targ;
    uintptr_t x,y,i;
    uintptr_t clock = 0;
    uintptr_t outputBuf[POND_DEPTH_SYSWORDS];
    uintptr_t currentWord,wordPtr,shiftPtr,inst,tmp;
    struct Cell *pptr,*tmpptr;
    uintptr_t ptr_wordPtr;
    uintptr_t ptr_shiftPtr;
    uintptr_t reg;
    uintptr_t facing;
    uintptr_t loopStack_wordPtr[POND_DEPTH];
    uintptr_t loopStack_shiftPtr[POND_DEPTH];
    uintptr_t loopStackPtr;
    uintptr_t falseLoopDepth;
    int stop;
    int skip;
    int access_neg_used;
    int access_pos_used;
    uintptr_t access_neg;
    uintptr_t access_pos;
    uintptr_t rand;
    if (!(clock % INFLOW_FREQUENCY)) {
        getRandomRollback(1, &x, buffer, in, prngState);
        x = x % POND_SIZE_X;
        getRandomRollback(1, &y, buffer, in, prngState);
        y = y % POND_SIZE_Y;
        pptr = &pond[y * POND_SIZE_X + x]; 
        pptr->ID = cellIdCounter;
        pptr->parentID = 0;
        pptr->lineage = cellIdCounter;
        pptr->generation = 0;
#ifdef INFLOW_RATE_VARIATION
        getRandomRollback(1, &rand, buffer, in, prngState);
        pptr->energy += INFLOW_RATE_BASE + (rand % INFLOW_RATE_VARIATION);
#else
        pptr->energy += INFLOW_RATE_BASE;
#endif /* INFLOW_RATE_VARIATION */
for(i=0;i<POND_DEPTH_SYSWORDS;++i) 
            getRandomRollback(1, &rand, buffer, in, prngState);
            pptr->genome[i] = rand;
        ++cellIdCounter;
    }
    /* Pick a random cell to execute */
    getRandomRollback(1, &rand, buffer, in, prngState);
    //
    x = rand % POND_SIZE_X;
    y = ((rand / POND_SIZE_X) >> 1) % POND_SIZE_Y;
    pptr = &pond[y * POND_SIZE_X + x];
    /* Reset the state of the VM prior to execution */
    for(i=0;i<POND_DEPTH_SYSWORDS;++i)
        outputBuf[i] = ~((uintptr_t)0); /* ~0 == 0xfffff... */
    ptr_wordPtr = 0;
    ptr_shiftPtr = 0;
    reg = 0;
    loopStackPtr = 0;
    wordPtr = EXEC_START_WORD;
    shiftPtr = EXEC_START_BIT;
    facing = 0;
    falseLoopDepth = 0;
    stop = 0;
    skip=0;
    access_neg_used = 0;
    access_pos_used = 0;
    access_neg = 0;
    access_pos = 0;
    /* We use a currentWord buffer to hold the word we're
        * currently working on.  This speeds things up a bit
        * since it eliminates a pointer dereference in the
        * inner loop. We have to be careful to refresh this
        * whenever it might have changed... take a look at
        * the code. :) */
    currentWord = pptr->genome[0];
    /* Keep track of how many cells have been executed */
    statCounters.cellExecutions += 1.0;
    /* Core execution loop */
 while ((pptr->energy)&&(!stop)) {
        /* Get the next instruction */
        inst = (currentWord >> shiftPtr) & 0xf;
        skip=0;
        /* Randomly frob either the instruction or the register with a
            * probability defined by MUTATION_RATE. This introduces variation,
            * and since the variation is introduced into the state of the VM
            * it can have all manner of different effects on the end result of
            * replication: insertions, deletions, duplications of entire
            * ranges of the genome, etc. */
        getRandomRollback(1, &rand, buffer, in, prngState);
        if ((rand & 0xffffffff) < MUTATION_RATE) {
            getRandomRollback(1, &rand, buffer, in, prngState);
            tmp = rand; // Call getRandom() only once for speed 
            if (tmp & 0x80) // Check for the 8th bit to get random boolean 
                inst = tmp & 0xf; // Only the first four bits are used here 
            else reg = tmp & 0xf;
        }
        /* Each instruction processed costs one unit of energy */
        --pptr->energy;
        /* Execute the instruction */
        if (falseLoopDepth) {
            /* Skip forward to matching REP if we're in a false loop. */
            if (inst == 0x9) /* Increment false LOOP depth */
                ++falseLoopDepth;
            else if (inst == 0xa) /* Decrement on REP */
                --falseLoopDepth;
        } else {
            statCounters.instructionExecutions[inst] += 1.0;
        }
    }
    /* Copy outputBuf into neighbor if access is permitted and there
        * is energy there to make something happen. There is no need
        * to copy to a cell with no energy, since anything copied there
        * would never be executed and then would be replaced with random
        * junk eventually. See the seeding code in the main loop above. */ 
 if ((outputBuf[0] & 0xff) != 0xff) {
        getNeighbor(pond,x,y,facing, tmpptr);
        //printf("%lu\n", tmpptr->energy);
        if ((tmpptr->energy)) {
            accessAllowed(tmpptr,reg,0,1, &rand, buffer, in, prngState);
            if(rand) {
            /* Log it if we're replacing a viable cell */
            if (tmpptr->generation > 2)
                ++statCounters.viableCellsReplaced;
            tmpptr->ID = ++cellIdCounter;
            tmpptr->parentID = pptr->ID;
            tmpptr->lineage = pptr->lineage; /* Lineage is copied in offspring */
            tmpptr->generation = pptr->generation + 1;
            for(i=0;i<POND_DEPTH_SYSWORDS;++i)
                tmpptr->genome[i] = outputBuf[i];
            }
        }
    }
}

